#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include "rotate_rect_ops.h"

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__device__ inline void get_rotated_rect_bounding_box(const T* pts, int& leftMost, int& topMost,
  int& rightMost, int& bottomMost, const int width, const int height)
{
//  const T* P = pts;
//  leftMost = int(max(round(min(min(P[0],P[2]),min(P[4],P[6]))),0.0));
//  rightMost= int(min(round(max(max(P[0],P[2]),max(P[4],P[6]))),width-1.0));
//  topMost= int(max(round(min(min(P[1],P[3]),min(P[5],P[7]))),0.0));
//  bottomMost= int(min(round(max(max(P[1],P[3]),max(P[5],P[7]))),height-1.0));

  leftMost = int(max(min(min(pts[0], pts[2]), min(pts[4], pts[6])), 0.0));
  topMost = int(max(min(min(pts[1], pts[3]), min(pts[5], pts[7])), 0.0));
  rightMost = int(min(max(max(pts[0], pts[2]), max(pts[4], pts[6])) + 1, width - 1.0));
  bottomMost = int(min(max(max(pts[1], pts[3]), max(pts[5], pts[7])) + 1, height - 1.0));
}


template <typename T>
__global__ void RRoIPoolFForward(const int nthreads, const T* bottom_data,
    const float spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const T* bottom_rois, T* top_data, int* argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];

    T P[8];
    compute_roi_pool_pts(offset_bottom_rois, P, spatial_scale, pooled_height, pooled_width, ph, pw);

    int leftMost, topMost, rightMost, bottomMost;
    get_rotated_rect_bounding_box(P, leftMost, topMost, rightMost, bottomMost, width, height);

    T maxval = 0;
    int maxidx = -1;
    const T* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;

    T AB[2];
    AB[0] = P[0] - P[2];
    AB[1] = P[1] - P[3];  
    T ABAB = AB[0]*AB[0] +AB[1]*AB[1];
    T AC[2];
    AC[0] = P[4] - P[2];
    AC[1] = P[5] - P[3];
    T ACAC = AC[0]*AC[0] + AC[1]*AC[1];

    for (int hh = topMost; hh < bottomMost+1; ++hh) {
      for (int ww = leftMost; ww < rightMost+1; ++ww) {
        T AP[2];
        AP[0] = ww - P[2];
        AP[1] = hh - P[3];
        T ABAP = AB[0]*AP[0] + AB[1]*AP[1];
        T ACAP = AC[0]*AP[0] + AC[1]*AP[1];
        if ( ABAP >= 1e-3 && (ABAB - ABAP) > -1e-3 && ACAP >= 1e-3 && (ACAC - ACAP) > -1e-3 )
        {
          int bottom_index = hh * width + ww;
          if (offset_bottom_data[bottom_index] > maxval) 
          {
            maxval = offset_bottom_data[bottom_index];
            maxidx = bottom_index;
          }
        }
      }
    }
    top_data[index] = maxval;
    argmax_data[index] = maxidx;

    // T bin_size_h = static_cast<T>(roi_height)
    //                    / static_cast<T>(pooled_height);
    // T bin_size_w = static_cast<T>(roi_width)
    //                    / static_cast<T>(pooled_width);

    // int hstart = static_cast<int>(floor(static_cast<T>(ph)
    //                                     * bin_size_h));
    // int wstart = static_cast<int>(floor(static_cast<T>(pw)
    //                                     * bin_size_w));
    // int hend = static_cast<int>(ceil(static_cast<T>(ph + 1)
    //                                  * bin_size_h));
    // int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
    //                                  * bin_size_w));

    // // Add roi offsets and clip to input boundaries
    // hstart = min(max(hstart + roi_start_h, 0), height);
    // hend = min(max(hend + roi_start_h, 0), height);
    // wstart = min(max(wstart + roi_start_w, 0), width);
    // wend = min(max(wend + roi_start_w, 0), width);
    // bool is_empty = (hend <= hstart) || (wend <= wstart);

    // // Define an empty pooling region to be zero
    // T maxval = is_empty ? 0 : -FLT_MAX;
    // // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    // int maxidx = -1;
    // const T* offset_bottom_data =
    //     bottom_data + (roi_batch_ind * channels + c) * height * width;
    // for (int h = hstart; h < hend; ++h) {
    //   for (int w = wstart; w < wend; ++w) {
    //     int bottom_index = h * width + w;
    //     if (offset_bottom_data[bottom_index] > maxval) {
    //       maxval = offset_bottom_data[bottom_index];
    //       maxidx = bottom_index;
    //     }
    //   }
    // }
    // top_data[index] = maxval;
    // argmax_data[index] = maxidx;
  }
}

template <typename T>
__global__ void RRoIPoolFBackward(const int nthreads, const T* top_diff,
    const int* argmax_data, const int num_rois, const float spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];
    int bottom_offset = (roi_batch_ind * channels + c) * height * width;
    int top_offset    = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    T* offset_bottom_diff = bottom_diff + bottom_offset;
    const int* offset_argmax_data = argmax_data + top_offset;

    int argmax = offset_argmax_data[ph * pooled_width + pw];
    if (argmax != -1) {
      atomicAdd(
          offset_bottom_diff + argmax,
          static_cast<T>(offset_top_diff[ph * pooled_width + pw]));

    }
  }
}

std::tuple<at::Tensor, at::Tensor> RROIPool_forward_cuda(const at::Tensor& input,
                                const at::Tensor& rois,
                                const float spatial_scale,
                                const int pooled_height,
                                const int pooled_width) {
  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.type().is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::empty({num_rois, channels, pooled_height, pooled_width}, input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
  auto argmax = at::zeros({num_rois, channels, pooled_height, pooled_width}, input.options().dtype(at::kInt));

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)output_size, 512L), 4096L));
  dim3 block(512);

  if (output.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return std::make_tuple(output, argmax);
  }

  AT_DISPATCH_FLOATING_TYPES(input.type(), "RROIPool_forward", [&] {
    RRoIPoolFForward<scalar_t><<<grid, block, 0, stream>>>(
         output_size,
         input.contiguous().data<scalar_t>(),
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         rois.contiguous().data<scalar_t>(),
         output.data<scalar_t>(),
         argmax.data<int>());
  });
  THCudaCheck(hipGetLastError());
  return std::make_tuple(output, argmax);
}

// TODO remove the dependency on input and use instead its sizes -> save memory
at::Tensor RROIPool_backward_cuda(const at::Tensor& grad,
                                 const at::Tensor& input,
                                 const at::Tensor& rois,
                                 const at::Tensor& argmax,
                                 const float spatial_scale,
                                 const int pooled_height,
                                 const int pooled_width,
                                 const int batch_size,
                                 const int channels,
                                 const int height,
                                 const int width) {
  AT_ASSERTM(grad.type().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.type().is_cuda(), "rois must be a CUDA tensor");
  // TODO add more checks

  auto num_rois = rois.size(0);
  auto grad_input = at::zeros({batch_size, channels, height, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)grad.numel(), 512L), 4096L));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return grad_input;
  }

  AT_DISPATCH_FLOATING_TYPES(grad.type(), "RROIPool_backward", [&] {
    RRoIPoolFBackward<scalar_t><<<grid, block, 0, stream>>>(
         grad.numel(),
         grad.contiguous().data<scalar_t>(),
         argmax.data<int>(),
         num_rois,
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         grad_input.data<scalar_t>(),
         rois.contiguous().data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return grad_input;
}
